#include "hip/hip_runtime.h"
// Copyright (c) 2024 SparseEnd2End. All rights reserved @author: Thomas Von Wu.
#include "img_aug_with_bilinearinterpolation_kernel.h"
namespace sparse_end2end {
namespace preprocessor {

#define R_MEAN 0.485F
#define G_MEAN 0.456F
#define B_MEAN 0.406F
#define R_STD 0.229F
#define G_STD 0.224F
#define B_STD 0.225F

#define DIVUP(a, b) ((a % b != 0) ? (a / b + 1) : (a / b))

// image prepocessor(resize+crop+bilinear_interpolation+normalization) CUDA kernel with output dtype: fp32.
__global__ void imgAugKernel(const std::uint8_t* input_ptr,  /// raw_imgs_cuda_ptr
                             const std::uint32_t n,          /// num_cams
                             const std::uint32_t c,          /// raw_img_c, target_img_c
                             const std::uint32_t h,          /// raw_img_h
                             const std::uint32_t w,          /// raw_img_w
                             const std::uint32_t target_h,   /// model_input_img_h
                             const std::uint32_t target_w,   /// model_input_img_w
                             const float resize_ratio,
                             const std::uint32_t crop_height,
                             const std::uint32_t crop_width,
                             float* output_ptr  /// model_input_imgs_cuda_ptr
) {
  const std::int32_t cam_id = blockIdx.x;
  const std::int32_t dst_y = blockIdx.y * blockDim.x + threadIdx.x;
  const std::int32_t dst_x = blockIdx.z * blockDim.y + threadIdx.y;

  if (dst_y >= target_h || dst_x >= target_w) {
    return;
  }

  const float resize_ratio_x = static_cast<float>(w) / static_cast<float>(std::floor(w * resize_ratio));
  const float resize_ratio_y = static_cast<float>(h) / static_cast<float>(std::floor(h * resize_ratio));

  const float src_x = (dst_x + crop_width + 0.5F) * resize_ratio_x - 0.5F;
  const float src_y = (dst_y + crop_height + 0.5F) * resize_ratio_y - 0.5F;

  std::uint32_t low_x = std::floor(src_x);
  std::uint32_t low_y = std::floor(src_y);

  std::uint32_t high_x = min(low_x + 1U, w - 1U);
  std::uint32_t high_y = min(low_y + 1U, h - 1U);

  low_x = max(0U, low_x);
  low_y = max(0U, low_y);

  const float ly = src_y - low_y;
  const float lx = src_x - low_x;
  const float hy = 1.0F - ly;
  const float hx = 1.0F - lx;

  const float w1 = hy * hx, w2 = hy * lx, w3 = ly * hx, w4 = ly * lx;

  const float value1_r = float(input_ptr[cam_id * (c * h * w) + 0 * (h * w) + low_y * w + low_x]);
  const float value1_g = float(input_ptr[cam_id * (c * h * w) + 1 * (h * w) + low_y * w + low_x]);
  const float value1_b = float(input_ptr[cam_id * (c * h * w) + 2 * (h * w) + low_y * w + low_x]);

  const float value2_r = float(input_ptr[cam_id * (c * h * w) + 0 * (h * w) + low_y * w + high_x]);
  const float value2_g = float(input_ptr[cam_id * (c * h * w) + 1 * (h * w) + low_y * w + high_x]);
  const float value2_b = float(input_ptr[cam_id * (c * h * w) + 2 * (h * w) + low_y * w + high_x]);

  const float value3_r = float(input_ptr[cam_id * (c * h * w) + 0 * (h * w) + high_y * w + low_x]);
  const float value3_g = float(input_ptr[cam_id * (c * h * w) + 1 * (h * w) + high_y * w + low_x]);
  const float value3_b = float(input_ptr[cam_id * (c * h * w) + 2 * (h * w) + high_y * w + low_x]);

  const float value4_r = float(input_ptr[cam_id * (c * h * w) + 0 * (h * w) + high_y * w + high_x]);
  const float value4_g = float(input_ptr[cam_id * (c * h * w) + 1 * (h * w) + high_y * w + high_x]);
  const float value4_b = float(input_ptr[cam_id * (c * h * w) + 2 * (h * w) + high_y * w + high_x]);

  float r_value = value1_r * w1 + value2_r * w2 + value3_r * w3 + value4_r * w4;
  float g_value = value1_g * w1 + value2_g * w2 + value3_g * w3 + value4_g * w4;
  float b_value = value1_b * w1 + value2_b * w2 + value3_b * w3 + value4_b * w4;

  r_value = r_value / 255.0F;
  g_value = g_value / 255.0F;
  b_value = b_value / 255.0F;

  r_value = (r_value - R_MEAN) / R_STD;
  g_value = (g_value - G_MEAN) / G_STD;
  b_value = (b_value - B_MEAN) / B_STD;

  output_ptr[cam_id * c * target_h * target_w + 0U * target_h * target_w + dst_y * target_w + dst_x] = r_value;
  output_ptr[cam_id * c * target_h * target_w + 1U * target_h * target_w + dst_y * target_w + dst_x] = g_value;
  output_ptr[cam_id * c * target_h * target_w + 2U * target_h * target_w + dst_y * target_w + dst_x] = b_value;
}

// image prepocessor(resize+crop+bilinear_interpolation+normalization) CUDA kernel with output dtype: fp16.
__global__ void imgAugKernel(const std::uint8_t* input_ptr,
                             const std::uint32_t n,
                             const std::uint32_t c,
                             const std::uint32_t h,
                             const std::uint32_t w,
                             const std::uint32_t target_h,
                             const std::uint32_t target_w,
                             const float resize_ratio,
                             const std::uint32_t crop_height,
                             const std::uint32_t crop_width,
                             half* output_ptr) {
  const std::int32_t cam_id = blockIdx.x;
  const std::int32_t dst_y = blockIdx.y * blockDim.x + threadIdx.x;
  const std::int32_t dst_x = blockIdx.z * blockDim.y + threadIdx.y;

  if (dst_y >= target_h || dst_x >= target_w) {
    return;
  }

  const float resize_ratio_x = static_cast<float>(w) / static_cast<float>(std::floor(w * resize_ratio));
  const float resize_ratio_y = static_cast<float>(h) / static_cast<float>(std::floor(h * resize_ratio));

  const float src_x = (dst_x + crop_width + 0.5F) * resize_ratio_x - 0.5F;
  const float src_y = (dst_y + crop_height + 0.5F) * resize_ratio_y - 0.5F;

  std::uint32_t low_x = std::floor(src_x);
  std::uint32_t low_y = std::floor(src_y);

  std::uint32_t high_x = min(low_x + 1U, w - 1U);
  std::uint32_t high_y = min(low_y + 1U, h - 1U);

  low_x = max(0U, low_x);
  low_y = max(0U, low_y);

  const float ly = src_y - low_y;
  const float lx = src_x - low_x;
  const float hy = 1.0F - ly;
  const float hx = 1.0F - lx;

  const float w1 = hy * hx, w2 = hy * lx, w3 = ly * hx, w4 = ly * lx;

  const float value1_r = float(input_ptr[cam_id * (c * h * w) + 0 * (h * w) + low_y * w + low_x]);
  const float value1_g = float(input_ptr[cam_id * (c * h * w) + 1 * (h * w) + low_y * w + low_x]);
  const float value1_b = float(input_ptr[cam_id * (c * h * w) + 2 * (h * w) + low_y * w + low_x]);

  const float value2_r = float(input_ptr[cam_id * (c * h * w) + 0 * (h * w) + low_y * w + high_x]);
  const float value2_g = float(input_ptr[cam_id * (c * h * w) + 1 * (h * w) + low_y * w + high_x]);
  const float value2_b = float(input_ptr[cam_id * (c * h * w) + 2 * (h * w) + low_y * w + high_x]);

  const float value3_r = float(input_ptr[cam_id * (c * h * w) + 0 * (h * w) + high_y * w + low_x]);
  const float value3_g = float(input_ptr[cam_id * (c * h * w) + 1 * (h * w) + high_y * w + low_x]);
  const float value3_b = float(input_ptr[cam_id * (c * h * w) + 2 * (h * w) + high_y * w + low_x]);

  const float value4_r = float(input_ptr[cam_id * (c * h * w) + 0 * (h * w) + high_y * w + high_x]);
  const float value4_g = float(input_ptr[cam_id * (c * h * w) + 1 * (h * w) + high_y * w + high_x]);
  const float value4_b = float(input_ptr[cam_id * (c * h * w) + 2 * (h * w) + high_y * w + high_x]);

  float r_value = value1_r * w1 + value2_r * w2 + value3_r * w3 + value4_r * w4;
  float g_value = value1_g * w1 + value2_g * w2 + value3_g * w3 + value4_g * w4;
  float b_value = value1_b * w1 + value2_b * w2 + value3_b * w3 + value4_b * w4;

  r_value = r_value / 255.0F;
  g_value = g_value / 255.0F;
  b_value = b_value / 255.0F;

  r_value = (r_value - R_MEAN) / R_STD;
  g_value = (g_value - G_MEAN) / G_STD;
  b_value = (b_value - B_MEAN) / B_STD;

  output_ptr[cam_id * c * target_h * target_w + 0U * target_h * target_w + dst_y * target_w + dst_x] =
      __float2half(r_value);
  output_ptr[cam_id * c * target_h * target_w + 1U * target_h * target_w + dst_y * target_w + dst_x] =
      __float2half(g_value);
  output_ptr[cam_id * c * target_h * target_w + 2U * target_h * target_w + dst_y * target_w + dst_x] =
      __float2half(b_value);
}

common::Status imgPreprocessLauncher(const std::uint8_t* raw_imgs_cuda_ptr,
                                     const std::uint32_t& num_cams,
                                     const std::uint32_t& raw_img_c,
                                     const std::uint32_t& raw_img_h,
                                     const std::uint32_t& raw_img_w,
                                     const std::uint32_t& model_input_img_h,
                                     const std::uint32_t& model_input_img_w,
                                     const float& resize_ratio,
                                     const std::uint32_t& crop_height,
                                     const std::uint32_t& crop_width,
                                     const hipStream_t& stream,
                                     float* model_input_imgs_cuda_ptr)

{
  const std::uint32_t thread_num = 32U;
  dim3 blocks_dim_in_each_grid(num_cams, DIVUP(model_input_img_h, thread_num), DIVUP(model_input_img_w, thread_num));
  dim3 threads_dim_in_each_block(thread_num, thread_num);

  imgAugKernel<<<blocks_dim_in_each_grid, threads_dim_in_each_block, 0, stream>>>(
      raw_imgs_cuda_ptr, num_cams, raw_img_c, raw_img_h, raw_img_w, model_input_img_h, model_input_img_w, resize_ratio,
      crop_height, crop_width, model_input_imgs_cuda_ptr);

  if (hipError_t::hipSuccess == hipGetLastError()) {
    return common::Status::kSuccess;
  } else {
    return common::Status::kImgPreprocessLaunchErr;
  }
}

common::Status imgPreprocessLauncher(const std::uint8_t* raw_imgs_cuda_ptr,
                                     const std::uint32_t& num_cams,
                                     const std::uint32_t& raw_img_c,
                                     const std::uint32_t& raw_img_h,
                                     const std::uint32_t& raw_img_w,
                                     const std::uint32_t& model_input_img_h,
                                     const std::uint32_t& model_input_img_w,
                                     const float& resize_ratio,
                                     const std::uint32_t& crop_height,
                                     const std::uint32_t& crop_width,
                                     const hipStream_t& stream,
                                     half* model_input_imgs_cuda_ptr)

{
  const std::uint32_t thread_num = 32U;
  dim3 blocks_dim_in_each_grid(num_cams, DIVUP(model_input_img_h, thread_num), DIVUP(model_input_img_w, thread_num));
  dim3 threads_dim_in_each_block(thread_num, thread_num);

  imgAugKernel<<<blocks_dim_in_each_grid, threads_dim_in_each_block, 0, stream>>>(
      raw_imgs_cuda_ptr, num_cams, raw_img_c, raw_img_h, raw_img_w, model_input_img_h, model_input_img_w, resize_ratio,
      crop_height, crop_width, model_input_imgs_cuda_ptr);

  if (hipError_t::hipSuccess == hipGetLastError()) {
    return common::Status::kSuccess;
  } else {
    return common::Status::kImgPreprocessLaunchErr;
  }
}

}  // namespace preprocessor
}  // namespace sparse_end2end
